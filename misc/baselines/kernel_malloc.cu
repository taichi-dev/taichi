#include "hip/hip_runtime.h"
#include <cstdio>
#include "time.h"

constexpr int segment_size = 1024;
constexpr int threads = 512;
__device__ char *pool;

void __global__ alloc(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  // pointers[index] = (int *)malloc(segment_size);
  pointers[index] = (int *)atomicAdd((unsigned long long *)&pool, segment_size);
}

void __global__ fill(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = 0; i < segment_size / sizeof(int); i++) {
    pointers[index][i] = i;
  }
}

void __global__ free(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  // free(pointers[index]);
}

int main() {
  int **pointers;
  hipMalloc(&pointers, threads * sizeof(int *));

  int bd = 32;
  for (int i = 0; i < 10; i++) {
    char *pool_;
    hipMallocManaged(&pool_, segment_size * threads);
    hipMemcpyToSymbol(HIP_SYMBOL(pool), &pool_, sizeof(void *));
    alloc<<<threads / bd, bd>>>(pointers);
    fill<<<threads / bd, bd>>>(pointers);
    free<<<threads / bd, bd>>>(pointers);
  }
  hipDeviceSynchronize();
}
