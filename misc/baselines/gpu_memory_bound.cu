#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "get_time.h"

__global__ void cpy(float *a, float *b, int *c, int n) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = b[i];
}

int main() {
  int n = 1024 * 1024 * 1024 / 4;
  float *a, *b;
  int *c;
  hipMalloc(&a, n * sizeof(float));
  hipMalloc(&b, n * sizeof(float));
  hipMalloc(&c, n * sizeof(float));
  for (auto bs : {16, 32, 64, 128, 256}) {
    for (int i = 0; i < 10; i++) {
      cpy<<<n / bs, bs>>>(a, b, c, n);
    }
    hipDeviceSynchronize();
    int repeat = 100;
    auto t = get_time();
    for (int i = 0; i < repeat; i++) {
      cpy<<<n / bs, bs>>>(a, b, c, n);
    }
    hipDeviceSynchronize();
    t = (get_time() - t) / repeat;
    printf("memcpy 1GB data, block_size %d, %.2f ms   bw %.3f GB/s\n", bs,
           t * 1000, n * 8.0 / t / (1024 * 1024 * 1024.0f));
  }
}
